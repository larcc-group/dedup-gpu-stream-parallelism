#include "memutils.h"
#include "cudautils.h"
void allocateGpu(void ** pointer, size_t size){
    CUDA_SAFE_CALL(hipMalloc(pointer, size));
}
void setDevice(int index){
    hipSetDevice(index);
}
void freeGpu(void * pointer){
    hipFree(pointer);
}
void moveToGpu(void * dst, void * src, size_t size){
    CUDA_SAFE_CALL(hipMemcpyAsync(dst,src,size, hipMemcpyHostToDevice));
}
void startGpu(){
    hipSetDevice(0);
}